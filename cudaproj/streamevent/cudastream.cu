#include <iostream>
#include <hip/hip_runtime.h>

// Kernel functions to perform computation
__global__ void kernel1(int64_t *data, int64_t repeat) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (size_t i = 0; i < repeat; i++)
    {
        data[idx] += 1;
    }
}

__global__ void kernel2(int64_t *data, int64_t repeat) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (size_t i = 0; i < repeat; i++)
    {
        data[idx] += 2;
    }
}

__global__ void kernel3(int64_t *data, int64_t repeat) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (size_t i = 0; i < repeat; i++)
    {
        data[idx] -= 1;
    }
}

int main() {
    const int dataSize = 1024;
    const int printSize = 10;
    int64_t *h_data = new int64_t[dataSize]; // Host data
    int64_t *d_data1, *d_data2; // Device data

    // Initialize host data
    for (int i = 0; i < dataSize; i++) {
        h_data[i] = 0;
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_data1, dataSize * sizeof(int64_t));
    hipMalloc((void**)&d_data2, dataSize * sizeof(int64_t));

    // Transfer data from host to device
    hipMemcpy(d_data1, h_data, dataSize * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_data2, h_data, dataSize * sizeof(int64_t), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(256);
    dim3 gridDim((dataSize + blockDim.x - 1) / blockDim.x);

    // Create streams and event
    hipStream_t stream1, stream2;
    hipEvent_t event1;
    int priorityHigh, priorityLow;
    hipDeviceGetStreamPriorityRange(&priorityLow, &priorityHigh);
    hipStreamCreate(&stream1);
    hipStreamCreateWithPriority(&stream2, hipStreamDefault, priorityHigh);
    hipEventCreate(&event1);

    const int64_t repeat = 1000;

    // Execute kernel1 in stream1
    kernel1<<<gridDim, blockDim, 0, stream1>>>(d_data1, repeat);
    hipEventRecord(event1, stream1); // Record event1 after kernel1 execution in stream1

    // Execute kernel2 in stream2, waiting for event1
    hipStreamWaitEvent(stream2, event1, 0);

    kernel2<<<gridDim, blockDim, 0, stream2>>>(d_data1, repeat);

    // Execute kernel3 in stream1 on a different array
    kernel3<<<gridDim, blockDim, 0, stream1>>>(d_data2, repeat);
    // Synchronize streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Transfer data back from device to host
    hipMemcpy(h_data, d_data1, dataSize * sizeof(int64_t), hipMemcpyDeviceToHost);

    // Display the result for d_data1
    std::cout << "Data after kernel1 and kernel2:" << std::endl;
    for (int i = 0; i < printSize; i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    // Transfer data back from device to host for d_data2
    hipMemcpy(h_data, d_data2, dataSize * sizeof(int64_t), hipMemcpyDeviceToHost);

    // Display the result for d_data2
    std::cout << "Data after kernel3:" << std::endl;
    for (int i = 0; i < printSize; i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory and destroy streams and event
    hipFree(d_data1);
    hipFree(d_data2);
    delete[] h_data;
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(event1);

    return 0;
}